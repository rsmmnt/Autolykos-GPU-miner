// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolykos puzzle cycle

*******************************************************************************/
#include "../include/websocket.h"
#include "../include/compaction.h"
#include "../include/conversion.h"
#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/easylogging++.h"
#include "../include/jsmn.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/processing.h"
#include "../include/reduction.h"
#include "../include/request.h"
#include <atomic>
#include <chrono>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <iostream>
#include <mutex>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <thread>
#include <vector>

#ifdef COMM_WEBSOCKET
#include "../include/websocket.h"
#endif

#ifdef _WIN32
#include <io.h>
#define R_OK    4       
#define W_OK    2       
#define F_OK    0       
#define access _access
#else
#include <unistd.h>
#endif

INITIALIZE_EASYLOGGINGPP

using namespace std::chrono;

void MinerThread(int deviceId, info_t * info);

////////////////////////////////////////////////////////////////////////////////
//  Main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char ** argv)
{
    START_EASYLOGGINGPP(argc, argv);

    el::Loggers::reconfigureAllLoggers(
        el::ConfigurationType::Format, "%datetime %level [%thread] %msg"
    );

    el::Helpers::setThreadName("main thread");

    int deviceCount;
    int status = EXIT_SUCCESS;

    info_t info;
    info.blockId = 1;
    info.keepPrehash = 0;

    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        LOG(ERROR) << "Error checking GPU";
        return EXIT_FAILURE;
    }

    LOG(INFO) << "Using " << deviceCount << " GPU devices";

    PERSISTENT_CALL_STATUS(curl_global_init(CURL_GLOBAL_ALL), CURLE_OK);

    char confName[14] = "./config.json";
    char * fileName = (argc == 1)? confName: argv[1];
    char from[MAX_URL_SIZE];
    int diff;
    
    json_t request(0, REQ_LEN);
    
    LOG(INFO) << "Using configuration file " << fileName;


    // check access to config file
    if (access(fileName, F_OK) == -1)
    {
        LOG(ERROR) << "Config file " << fileName << " not found";
        return EXIT_FAILURE;
    }

    // read config from file
    status = ReadConfig(
        fileName, info.sk_h, info.skstr, from, info.to, &info.keepPrehash
    );

    if (status == EXIT_FAILURE)
    {
        LOG(ERROR) << "Wrong config file format";
        return EXIT_FAILURE;
    }

    LOG(INFO) << "Block getting URL " << from;
    LOG(INFO) << "Solution posting URL " << info.to;

    // generate public key from secret key
    GeneratePublicKey(info.skstr, info.pkstr, info.pk_h);
    
    char logstr[1000];

    sprintf(logstr,
        "Generated public key:\n"
        "   pk = 0x%02lX %016lX %016lX %016lX %016lX",
        ((uint8_t *)info.pk_h)[0],
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 0),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 1),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 2),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 3)
    );

    LOG(INFO) << logstr;
    #ifdef COMM_WEBSOCKET
    WebSocketComm wsComm("ws://localhost:3000", &info);



    #else
    status = GetLatestBlock(
        from, &request, &info, true
    );
    if(status != EXIT_SUCCESS)
    {
        LOG(INFO) << "First block getting request failed, maybe wrong node address?";
    }
    #endif

    std::vector<std::thread> miners(deviceCount);

    for (int i = 0; i < deviceCount; ++i)
    {
        miners[i] = std::thread(MinerThread, i, &info);
    }

    //====================================================================//
    //  Main cycle
    //====================================================================//
    // bomb node with HTTP with 10ms intervals, if new block came 
    // signal miners with blockId
    #ifdef COMM_WEBSOCKET
    
    while(1)
    {
        int rslt = wsComm.check();
        std::this_thread::sleep_for(std::chrono::milliseconds(8));


    }




    #else

    
    uint_t curlcnt = 0;
    const uint_t curltimes = 2000;

    // using namespace std::chrono;
    milliseconds ms = milliseconds::zero(); 

    while(!TerminationRequestHandler())
    {
        milliseconds start = duration_cast<milliseconds>(
            system_clock::now().time_since_epoch()
        );
        
        status = GetLatestBlock(
            from, &request, &info, false);
        
        if (status != EXIT_SUCCESS) { LOG(INFO) << "Getting block error"; }

        ms += duration_cast<milliseconds>(
            system_clock::now().time_since_epoch()
        ) - start;

        ++curlcnt;

        if (!(curlcnt % curltimes))
        {
            LOG(INFO) << "Average curling time "
                << ms.count() / (double)curltimes << " ms";
            ms = milliseconds::zero();
        }

        std::this_thread::sleep_for(std::chrono::milliseconds(8));
    }    
    #endif
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//  Miner thread cycle
////////////////////////////////////////////////////////////////////////////////
void MinerThread(int deviceId, info_t * info)
{
    state_t state = STATE_KEYGEN;
    char threadName[20];

    CUDA_CALL(hipSetDevice(deviceId));
    sprintf(threadName, "GPU %i miner", deviceId);
    el::Helpers::setThreadName(threadName);    


    //====================================================================//
    //  Host memory allocation
    //====================================================================//
    // curl http request
    json_t request(0, REQ_LEN);

    // hash context
    // (212 + 4) bytes
    ctx_t ctx_h;

    // autolykos variables
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    uint8_t x_h[NUM_SIZE_8];
    uint8_t w_h[PK_SIZE_8];
    uint8_t res_h[NUM_SIZE_8];
    uint8_t nonce[NONCE_SIZE_8];

    // cryptography variables
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4 + 1];
    //char from[MAX_URL_SIZE];
    char to[MAX_URL_SIZE];
    int keepPrehash = 0;

    // thread info variables
    uint_t blockId = 0;
    milliseconds start; 
    
    //====================================================================//
    //  Copy from global to thread local data
    //====================================================================//
    info->info_mutex.lock();

    memcpy(sk_h, info->sk_h, NUM_SIZE_8);
    memcpy(mes_h, info->mes_h, NUM_SIZE_8);
    memcpy(bound_h, info->bound_h, NUM_SIZE_8);
    memcpy(pk_h, info->pk_h, PK_SIZE_8);
    memcpy(pkstr, info->pkstr, (PK_SIZE_4 + 1) * sizeof(char));
    memcpy(skstr, info->skstr, NUM_SIZE_4 * sizeof(char));
    memcpy(to, info->to, MAX_URL_SIZE * sizeof(char));
    // blockId = info->blockId.load();
    keepPrehash = info->keepPrehash;
    
    info->info_mutex.unlock();
    
    //====================================================================//
    //  Check GPU memory
    //====================================================================//
    size_t freeMem, totalMem;

    CUDA_CALL(hipMemGetInfo(&freeMem,&totalMem));
    
    if(freeMem < MIN_FREE_MEMORY)
    {
        LOG(ERROR) << "Not enough free GPU memory for mining, minimum 2.8 GiB needed";
        return;
    }

    if(keepPrehash && freeMem < MIN_FREE_MEMORY_PREHASH)
    {
        LOG(ERROR) << "Not enough free GPU memory for keeping prehashes, "
                   << "setting keepPrehash to false";
        keepPrehash = 0;
    }
    





    //====================================================================//
    //  Device memory allocation
    //====================================================================//
    LOG(INFO) << "GPU " << deviceId << " allocating memory";

    // boundary for puzzle
    // ~0 MiB
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc((void **)&bound_d, NUM_SIZE_8));

    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, DATA_SIZE_8));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc((void **)&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8 bytes // ~512 MiB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc(
        (void **)&indices_d, (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8
    ));

    // potential solutions of puzzle
    // THREAD_LEN * LOAD_LEN * NUM_SIZE_8 bytes // 128 MiB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, THREAD_LEN * LOAD_LEN * NUM_SIZE_8));

    // unfinalized hash contexts
    // N_LEN * 80 bytes // 5 GiB
    uctx_t * uctxs_d;

    if (keepPrehash)
    {
        CUDA_CALL(hipMalloc(
            (void **)&uctxs_d, (uint32_t)N_LEN * sizeof(uctx_t)
        ));
    }

    //====================================================================//
    //  Key-pair transfer form host to device
    //====================================================================//
    // copy public key
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, PK_SIZE_8, hipMemcpyHostToDevice
    ));

    // copy secret key
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + COUPLED_PK_SIZE_32 + 2 * NUM_SIZE_32), (void *)sk_h,
        NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    //int diff = 0;
    uint32_t ind = 0;
    uint64_t base = 0;

    if (keepPrehash)
    {
        LOG(INFO) << "Preparing unfinalized hashes on GPU " << deviceId;

        UncompleteInitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            data_d, uctxs_d
        );

        CUDA_CALL(hipDeviceSynchronize());
    }

    int cntCycles = 0;
    int NCycles = 100;
    start = duration_cast<milliseconds>(system_clock::now().time_since_epoch());

    do
    {
        ++cntCycles;

        if (!(cntCycles % NCycles))
        {
            milliseconds timediff
                = duration_cast<milliseconds>(
                    system_clock::now().time_since_epoch()
                ) - start;

            LOG(INFO) << "GPU " << deviceId << " hashrate "
                << (double)LOAD_LEN * NCycles
                / ((double)1000 * timediff.count()) << " MH/s";

            start = duration_cast<milliseconds>(
                system_clock::now().time_since_epoch()
            );
        }
    
        // if solution was found by this thread wait for new block to come 
        if (state == STATE_KEYGEN)
        {
            while (info->blockId.load() == blockId) {}

            state = STATE_CONTINUE;
        }

        uint_t controlId = info->blockId.load();

        if (blockId != controlId)
        {
            // if info->blockId changed
            // read new message and bound to thread-local mem
            info->info_mutex.lock();

            memcpy(mes_h, info->mes_h, NUM_SIZE_8);
            memcpy(bound_h, info->bound_h, NUM_SIZE_8);

            info->info_mutex.unlock();

            state = STATE_REHASH;
            LOG(INFO) << "GPU " << deviceId << " read new block data";
            blockId = controlId;
            
            GenerateKeyPair(x_h, w_h);

            VLOG(1) << "Generated new keypair,"
                << " copying new data in device memory now";

            // copy boundary
            CUDA_CALL(hipMemcpy(
                (void *)bound_d, (void *)bound_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));

            // copy message
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8), (void *)mes_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time secret key
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + COUPLED_PK_SIZE_32 + NUM_SIZE_32),
                (void *)x_h, NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time public key
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8),
                (void *)w_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));

            VLOG(1) << "Starting prehashing with new block data";
            Prehash(keepPrehash, data_d, uctxs_d, hashes_d, indices_d);
 
            state = STATE_CONTINUE;
        }

        CUDA_CALL(hipDeviceSynchronize());

        VLOG(1) << "Starting mining cycle";

        // restart iteration if new block was found
        if (blockId != info->blockId.load()) { continue; }

        // calculate unfinalized hash of message
        VLOG(1) << "Starting InitMining";
        InitMining(&ctx_h, (uint32_t *)mes_h, NUM_SIZE_8);

        // copy context
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + COUPLED_PK_SIZE_32 + 3 * NUM_SIZE_32),
            (void *)&ctx_h, sizeof(ctx_t), hipMemcpyHostToDevice
        ));

        // restart iteration if new block was found
        if (blockId != info->blockId.load()) { continue; }

        VLOG(1) << "Starting main BlockMining procedure";

        // calculate solution candidates
        BlockMining<<<1 + (LOAD_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            bound_d, data_d, base, hashes_d, res_d, indices_d
        );

        VLOG(1) << "Trying to find solution";

        // restart iteration if new block was found
        if (blockId != info->blockId.load()) { continue; }

        // try to find solution
        ind = FindNonZero(
            indices_d, indices_d + THREAD_LEN * LOAD_LEN, THREAD_LEN * LOAD_LEN
        );

        // solution found
        if (ind)
        {
            CUDA_CALL(hipMemcpy(
                (void *)res_h, (void *)(res_d + ((ind - 1) << 3)), NUM_SIZE_8,
                hipMemcpyDeviceToHost
            ));

            *((uint64_t *)nonce) = base + ind - 1;

            PrintPuzzleSolution(nonce, res_h);
            PostPuzzleSolution(to, pkstr, w_h, nonce, res_h);

            LOG(INFO) << "GPU " << deviceId << " found and posted a solution";
    
            state = STATE_KEYGEN;
        }

        base += THREAD_LEN * LOAD_LEN;
    }
    while(1); // !TerminationRequestHandler()); 

    return;
}

// autolykos.cu
