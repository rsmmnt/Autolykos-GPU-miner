// autolykos.cu

/*******************************************************************************

    AUTOLYKOS -- Autolykos puzzle cycle

*******************************************************************************/
#include "../include/easylogging++.h"
#include "../include/compaction.h"
#include "../include/conversion.h"
#include "../include/cryptography.h"
#include "../include/definitions.h"
#include "../include/jsmn.h"
#include "../include/mining.h"
#include "../include/prehash.h"
#include "../include/processing.h"
#include "../include/reduction.h"
#include "../include/request.h"
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <curl/curl.h>
#include <inttypes.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <time.h>
//#include <unistd.h>
#include <atomic>
#include <thread>
#include <chrono>
#include <mutex>
#include <vector>
#include <iostream>
#define TEXT_SEPARATOR   "========================================"\
                         "========================================\n"
#define TEXT_GPUCHECK    " Checking GPU availability\n"
#define TEXT_TERMINATION " Miner is now terminated\n"
#define ERROR_GPUCHECK   "ABORT:  GPU devices are not recognised\n"

INITIALIZE_EASYLOGGINGPP

using namespace std::chrono;


struct globalInfo
{

    // Mutex for reading/writing data from globalInfo safely
    std::mutex info_mutex;

    // Puzzle data to read
    
    	
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4 + 1];
    int keepPrehash;
    char to[MAX_URL_SIZE];
    

    // Increment when new block is sent by node

    std::atomic<unsigned int> blockId; 
};

void minerThread(int deviceId, globalInfo *info);


int main(int argc, char* argv[])
{
 
   START_EASYLOGGINGPP(argc, argv);
    el::Loggers::reconfigureAllLoggers(el::ConfigurationType::Format, "%datetime %level [%thread] %msg");
    el::Helpers::setThreadName("main thread");
    int deviceCount;
    timestamp_t stamp;
    int status = EXIT_SUCCESS;
    globalInfo info;
    info.blockId = 1;
    state_t state = STATE_CONTINUE;
    if (hipGetDeviceCount(&deviceCount) != hipSuccess)
    {
        /*
        fprintf(
            stderr, ERROR_GPUCHECK "%s" TEXT_TERMINATION TEXT_SEPARATOR,
            TimeStamp(&stamp)
        );
        */

        LOG(ERROR) << "Error checking GPU";

        return EXIT_FAILURE;
    }

    LOG(INFO) << "Using " << deviceCount <<" CUDA devices " ;
    //printf("Using %i CUDA devices\n",deviceCount);

    PERSISTENT_CALL_STATUS(curl_global_init(CURL_GLOBAL_ALL), CURLE_OK);
	

    char confname[14] = "./config.json";
    char * filename = (argc == 1)? confname: argv[1];
    char from[MAX_URL_SIZE];
    char to[MAX_URL_SIZE];
    int diff;
   // int keepPrehash = 0;
    json_t request(0, REQ_LEN);
    
    LOG(INFO) << "Using configuration file from " << filename ;

    /*
    printf(
        "Using configuration from \'%s\'\n", filename
    );
    fflush(stdout);
    */
    // check access to config file
    /*
    if (access(filename, F_OK) == -1)
    {
        /*
        fprintf(stderr, "ABORT:  File \'%s\' not found\n", filename);

        fprintf(
            stderr, "%s" TEXT_TERMINATION TEXT_SEPARATOR, TimeStamp(&stamp)
        );
        

        LOG(ERROR) << "Config file not found " << filename;

        return EXIT_FAILURE;
    }
    */
    // read config from file
    status = ReadConfig(filename, info.sk_h, info.skstr, from, info.to, &info.keepPrehash, &stamp);

    if (status == EXIT_FAILURE)
    {
        
        LOG(ERROR) << "Wrong config file format";
        /*fprintf(stderr, "ABORT:  Wrong config format\n");

        fprintf(
            stderr, "%s" TEXT_TERMINATION TEXT_SEPARATOR, TimeStamp(&stamp)
        );
        */
        return EXIT_FAILURE;
    }
    LOG(INFO) << "Block getting URL " << from;
    LOG(INFO) << "Solution postin URL " << info.to;
    // generate public key from secret key
    GeneratePublicKey(info.skstr, info.pkstr, info.pk_h);
    
    char logst[1000];

    sprintf(logst,
        "%s Generated public key:"
        "   pk = 0x%02lX %016lX %016lX %016lX %016lX",
        TimeStamp(&stamp), ((uint8_t *)info.pk_h)[0],
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 0),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 1),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 2),
        REVERSE_ENDIAN((uint64_t *)(info.pk_h + 1) + 3)
    );
    //fflush(stdout);
    LOG(INFO) << logst;

    status = GetLatestBlock(
        from, info.pkstr, &request, info.bound_h, info.mes_h, &state, &diff, true, info.info_mutex, info.blockId
    );
    if(status != EXIT_SUCCESS)
    {
        LOG(INFO) << "First block getting request failed, maybe wrong node address?";
    }


    std::vector<std::thread> miners(deviceCount);
    for(int i = 0; i < deviceCount; i++)
    {
        miners[i] = std::thread(minerThread, i, &info);

    }

    // main cycle - bomb node with HTTP with 10ms intervals, if new block came 
    //-> signal miners with blockId
    int curlcnt = 0;
    const int curltimes = 2000;
    //time_t differ = 0;

    //using namespace std::chrono;
    milliseconds ms = milliseconds::zero(); 

    while(!TerminationRequestHandler())
    {
        milliseconds start = duration_cast< milliseconds >(
            system_clock::now().time_since_epoch()
            );
        //info.info_mutex.lock();
        // need to fix state somehow
        state = STATE_CONTINUE;
        
        status = GetLatestBlock(
            from, info.pkstr, &request, info.bound_h, info.mes_h, &state, &diff, false, info.info_mutex, info.blockId);
        
        if(status != EXIT_SUCCESS)
	    {
            LOG(INFO) << "Getting block error";
            //printf("Getting block error\n");
	    }
        //info.info_mutex.unlock();

        ms +=  duration_cast< milliseconds >(system_clock::now().time_since_epoch()) - start;
        curlcnt++;
        if(curlcnt%curltimes == 0)
        {
            //printf("Average curling time %lf\n",(double)differ/(CLOCKS_PER_SEC*curltimes));
            LOG(INFO) << "Average curling time " << ms.count()/(double)curltimes << " ms";
            ms = milliseconds::zero();
        }
        /*
        if(diff || state == STATE_REHASH)
        {
            info.blockId++;
            diff = 0;
            LOG(INFO) << "Got new block in main thread"; 
            //printf("Got new block in main thread\n");
	        fflush(stdout);
        }
        */
        std::this_thread::sleep_for(std::chrono::milliseconds(8));

    }    


    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//  Main cycle
////////////////////////////////////////////////////////////////////////////////
void minerThread(int deviceId, globalInfo *info)
{
    int status = EXIT_SUCCESS;
    timestamp_t stamp;
    state_t state = STATE_KEYGEN;
    hipSetDevice(deviceId);
    char threadName[20];
    sprintf(threadName, "GPU %i miner",deviceId);
    el::Helpers::setThreadName(threadName);    

    //====================================================================//
    //  Host memory allocation
    //====================================================================//
    // curl http request
    json_t request(0, REQ_LEN);

    // hash context
    // (212 + 4) bytes
    context_t ctx_h;

    // autolykos variables
    uint8_t bound_h[NUM_SIZE_8];
    uint8_t mes_h[NUM_SIZE_8];
    uint8_t sk_h[NUM_SIZE_8];
    uint8_t pk_h[PK_SIZE_8];
    uint8_t x_h[NUM_SIZE_8];
    uint8_t w_h[PK_SIZE_8];
    uint8_t res_h[NUM_SIZE_8];
    uint8_t nonces_h[NONCE_SIZE_8];

    // cryptography variables
    char skstr[NUM_SIZE_4];
    char pkstr[PK_SIZE_4 + 1];
    char from[MAX_URL_SIZE];
    char to[MAX_URL_SIZE];
    int keepPrehash = 0;
    unsigned int blockId = 0;
    milliseconds start;	
    
    // Copy from global to thread local data
    //===============================================

    info->info_mutex.lock();

    memcpy(sk_h,info->sk_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(mes_h, info->mes_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(bound_h, info->bound_h, NUM_SIZE_8*sizeof(uint8_t));
    memcpy(pk_h, info->pk_h, PK_SIZE_8*sizeof(uint8_t));
    memcpy(pkstr, info->pkstr, (PK_SIZE_4+1)*sizeof(uint8_t));
    memcpy(skstr, info->skstr,NUM_SIZE_4*sizeof(uint8_t));
    memcpy(to, info->to, MAX_URL_SIZE*sizeof(char));
   // blockId = info->blockId.load();
    keepPrehash = info->keepPrehash;
    
    info->info_mutex.unlock();
    
    //end copy
    //===============================


    //====================================================================//
    //  Device memory allocation
    //====================================================================//
    //printf(" %s thread GPU %i allocating GPU memory\n", TimeStamp(&stamp), deviceId);
    LOG(INFO) << "GPU " << deviceId << " allocating memory";
    // fflush(stdout);

    // boundary for puzzle
    // ~0 MiB
    uint32_t * bound_d;
    CUDA_CALL(hipMalloc((void **)&bound_d, NUM_SIZE_8));

    // nonces
    // THREAD_LEN * LOAD_LEN * NONCE_SIZE_8 bytes // 32 MiB
    uint32_t * nonces_d;
    CUDA_CALL(hipMalloc(
        (void **)&nonces_d, THREAD_LEN * LOAD_LEN * NONCE_SIZE_8
    ));

    // data: pk || mes || w || padding || x || sk || ctx
    // (2 * PK_SIZE_8 + 2 + 3 * NUM_SIZE_8 + 212 + 4) bytes // ~0 MiB
    uint32_t * data_d;
    CUDA_CALL(hipMalloc((void **)&data_d, (NUM_SIZE_8 + BLOCK_DIM) * 4));

    // precalculated hashes
    // N_LEN * NUM_SIZE_8 bytes // 2 GiB
    uint32_t * hashes_d;
    CUDA_CALL(hipMalloc((void **)&hashes_d, (uint32_t)N_LEN * NUM_SIZE_8));

    // indices of unfinalized hashes
    // (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8 bytes // ~512 MiB
    uint32_t * indices_d;
    CUDA_CALL(hipMalloc(
        (void **)&indices_d, (THREAD_LEN * N_LEN * 2 + 1) * INDEX_SIZE_8
    ));

    // potential solutions of puzzle
    // THREAD_LEN * LOAD_LEN * NUM_SIZE_8 bytes // 128 MiB
    uint32_t * res_d;
    CUDA_CALL(hipMalloc((void **)&res_d, THREAD_LEN * LOAD_LEN * NUM_SIZE_8));

    // unfinalized hash contexts
    // N_LEN * 80 bytes // 5 GiB
    ucontext_type * uctxs_d;

    if (keepPrehash)
    {
        CUDA_CALL(hipMalloc(
            (void **)&uctxs_d, (uint32_t)N_LEN * sizeof(ucontext_type)
        ));
    }

    //====================================================================//
    //  Key-pair transfer form host to device
    //====================================================================//
    // copy public key
    CUDA_CALL(hipMemcpy(
        (void *)data_d, (void *)pk_h, PK_SIZE_8, hipMemcpyHostToDevice
    ));

    // copy secret key
    CUDA_CALL(hipMemcpy(
        (void *)(data_d + PK2_SIZE_32 + 2 * NUM_SIZE_32), (void *)sk_h,
        NUM_SIZE_8, hipMemcpyHostToDevice
    ));

    //====================================================================//
    //  Autolykos puzzle cycle
    //====================================================================//
    //state_t state = STATE_KEYGEN;
    int diff = 0;
    uint32_t ind = 0;
    uint64_t base = 0;

    if (keepPrehash)
    {
        /*
        printf(
            "%s Preparing unfinalized hashes\n" TEXT_SEPARATOR,
            TimeStamp(&stamp)
        );
        fflush(stdout);
        */
        LOG(INFO) << "Preparing unfinalized hashes on GPU " << deviceId;

        UncompleteInitPrehash<<<1 + (N_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            data_d, uctxs_d
        );

        CUDA_CALL(hipDeviceSynchronize());
    }

    int cntCycles = 0;
    int NCycles = 100;
    start = duration_cast<milliseconds> (system_clock::now().time_since_epoch());
    do
    {
        
	    cntCycles++;
	    if(cntCycles%NCycles == 0)
	    {
            milliseconds timediff = duration_cast<milliseconds> (system_clock::now().time_since_epoch()) - start;
            //printf("%lf MHashes per second on GPU %i \n", (double)LOAD_LEN*NCycles/((double)1000*timediff.count()), deviceId);
            LOG(INFO) << "GPU " << deviceId << " hashrate " << (double)LOAD_LEN*NCycles/((double)1000*timediff.count()) << " MH/s";
            start = duration_cast<milliseconds> (system_clock::now().time_since_epoch());
	    }
	
        // if solution was found by this thread, wait for new block to come 
        /*
        if(state == STATE_KEYGEN)
	    {
		    while(info->blockId.load() == blockId)
		    {}
		    state = STATE_CONTINUE;
	    }
        */
	    unsigned int controlId = info->blockId.load();
        if(blockId != controlId)
        {
            //if info->blockId changed, read new message and bound to thread-local mem

            info->info_mutex.lock();
            memcpy(mes_h, info->mes_h, NUM_SIZE_8*sizeof(uint8_t));
            memcpy(bound_h, info->bound_h, NUM_SIZE_8*sizeof(uint8_t));
            /*
            for(int i = 0; i < NUM_SIZE_8; i++)
            {
                mes_h[i] = info->mes_h[i];
                bound_h[i] = info->bound_h[i];
            }
            */
            info->info_mutex.unlock();
            state = STATE_REHASH;
	        //printf("Thread read new block data, blockid %i old %i\n",blockId,controlId);
            LOG(INFO) << "GPU " << deviceId << " read new block data";
            blockId = controlId;
            

            GenerateKeyPair(x_h, w_h);
        
            //PrintPuzzleState(mes_h, pk_h, sk_h, w_h, x_h, bound_h, &stamp);
            VLOG(1) << "Generated new keypair, copying new data in device memory now";
            // copy boundary
            CUDA_CALL(hipMemcpy(
                (void *)bound_d, (void *)bound_h, NUM_SIZE_8,
                hipMemcpyHostToDevice
            ));

            // copy message
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8), (void *)mes_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time secret key
            CUDA_CALL(hipMemcpy(
                (void *)(data_d + PK2_SIZE_32 + NUM_SIZE_32), (void *)x_h,
                NUM_SIZE_8, hipMemcpyHostToDevice
            ));

            // copy one time public key
            CUDA_CALL(hipMemcpy(
                (void *)((uint8_t *)data_d + PK_SIZE_8 + NUM_SIZE_8),
                (void *)w_h, PK_SIZE_8, hipMemcpyHostToDevice
            ));
            VLOG(1) << "Starting prehashing with new block data";
            Prehash(keepPrehash, data_d, uctxs_d, hashes_d, indices_d);
 

            state = STATE_CONTINUE;
    	    //printf("Prehashed for new block\n");
        }


        CUDA_CALL(hipDeviceSynchronize());
        VLOG(1) << "Starting mining cycle";
         /*     printf(
            "%s Checking solutions for nonces:\n"
            "           0x%016lX -- 0x%016lX\n",
            TimeStamp(&stamp), base, base + THREAD_LEN * LOAD_LEN - 1
        );
        fflush(stdout);
        */   
        // generate nonces
        GenerateConseqNonces<<<1 + (THREAD_LEN * LOAD_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            (uint64_t *)nonces_d, N_LEN, base
        );
        VLOG(1) << "Generating nonces";
        base += THREAD_LEN * LOAD_LEN;
        
        //interrupt cycle if new block was found
        if(blockId!=info->blockId.load())
	    {
		    continue;
	    }
        
        // calculate unfinalized hash of message
        VLOG(1) << "Starting InitMining";
        InitMining(&ctx_h, (uint32_t *)mes_h, NUM_SIZE_8);

        
        //interrupt cycle if new block was found
	    if(blockId!=info->blockId.load())
	    {
		    continue;
	    }

        // copy context
        CUDA_CALL(hipMemcpy(
            (void *)(data_d + PK2_SIZE_32 + 3 * NUM_SIZE_32), (void *)&ctx_h,
            sizeof(context_t), hipMemcpyHostToDevice
        ));
        VLOG(1) << "Starting main BlockMining procedure";
        // calculate solution candidates
        BlockMining<<<1 + (LOAD_LEN - 1) / BLOCK_DIM, BLOCK_DIM>>>(
            bound_d, data_d, nonces_d, hashes_d, res_d, indices_d
        );
        VLOG(1) << "Trying to find solution";
	    //interrupt cycle if new block was found
	    if(blockId!=info->blockId.load())
	    {
		    continue;
	    }
        // try to find solution
        ind = FindNonZero(
            indices_d, indices_d + THREAD_LEN * LOAD_LEN, THREAD_LEN * LOAD_LEN
        );

        // solution found
        if (ind)
        {
            CUDA_CALL(hipMemcpy(
                (void *)res_h, (void *)(res_d + ((ind - 1) << 3)), NUM_SIZE_8,
                hipMemcpyDeviceToHost
            ));

            CUDA_CALL(hipMemcpy(
                (void *)nonces_h, (void *)(nonces_d + ((ind - 1) << 1)),
                NONCE_SIZE_8, hipMemcpyDeviceToHost
            ));

            //printf("%s Solution found from GPU %i:\n", TimeStamp(&stamp), deviceId); 
            //PrintPuzzleSolution(nonces_h, res_h);
            PostPuzzleSolution(to, pkstr, w_h, nonces_h, res_h);
            LOG(INFO) << "GPU " << deviceId << " found and posted a solution";
            //printf("new Solution is posted\n");
            //fflush(stdout);
	
            state = STATE_KEYGEN;
        }
    }
    while(1); // !TerminationRequestHandler()); 

    return;
}

// autolykos.cu
