#include "hip/hip_runtime.h"
// mining.cu

/*******************************************************************************

    MINING -- Autolykos parallel BlockMining procedure

*******************************************************************************/

#include "../include/mining.h"
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//  Unfinalized hash of message
////////////////////////////////////////////////////////////////////////////////
void InitMining(
    // context
    context_t * ctx,
    // message
    const uint32_t * mes,
    // message length in bytes
    const uint32_t meslen
)
{
    int j;

    uint64_t aux[32];

    //====================================================================//
    //  Initialize context
    //====================================================================//
    memset(ctx->b, 0, BUF_SIZE_8);
    B2B_IV(ctx->h);
    ctx->h[0] ^= 0x01010000 ^ NUM_SIZE_8;
    memset(ctx->t, 0, 16);
    ctx->c = 0;

    //====================================================================//
    //  Hash message
    //====================================================================//
    for (j = 0; j < meslen; ++j)
    {
        if (ctx->c == BUF_SIZE_8)
        {
            HOST_B2B_H(ctx, aux);
        }

        ctx->b[ctx->c++] = ((const uint8_t *)mes)[j];
    }

    return;
}

////////////////////////////////////////////////////////////////////////////////
//  Block mining                                                               
////////////////////////////////////////////////////////////////////////////////
__global__ void BlockMining(
    // boundary for puzzle
    const uint32_t * bound,
    // data: pk || mes || w || padding || x || sk || ctx
    const uint32_t * data,
    // pregenerated nonces
    const uint32_t * non,
    // precalculated hashes
    const uint32_t * hash,
    // results
    uint32_t * res,
    // indices of valid solutions
    uint32_t * valid
)
{
    uint32_t j;
    uint32_t tid = threadIdx.x;

    // shared memory
    // BLOCK_DIM * 4 bytes  
    __shared__ uint32_t sdata[BLOCK_DIM];

    // BLOCK_DIM * 4 bytes
    sdata[tid] = data[tid + PK2_SIZE_32 + 2 * NUM_SIZE_32];
    __syncthreads();

    // NUM_SIZE_8 bytes
    uint32_t * sk = sdata;

    // local memory
    // 472 bytes
    uint32_t ldata[118];

    // 256 bytes
    uint64_t * aux = (uint64_t *)ldata;
    // (4 * K_LEN) bytes
    uint32_t * ind = ldata;
    // (NUM_SIZE_8 + 4) bytes
    uint32_t * r = ind + K_LEN;
    // (212 + 4) bytes 
    context_t * ctx = (context_t *)(ldata + 64);

#pragma unroll
    for (int l = 0; l < THREAD_LEN; ++l) 
    {
        *ctx = *((context_t *)(sdata + NUM_SIZE_32));

        tid = threadIdx.x + blockDim.x * blockIdx.x
            + l * gridDim.x * blockDim.x;

        const uint8_t * mes = (const uint8_t *)(non + tid * NONCE_SIZE_32);

    //====================================================================//
    //  Hash nonce
    //====================================================================//
#pragma unroll
        for (j = 0; ctx->c < BUF_SIZE_8 && j < NONCE_SIZE_8; ++j)
        {
            ctx->b[ctx->c++] = mes[j];
        }

#pragma unroll
        for ( ; j < NONCE_SIZE_8; )
        {
            DEVICE_B2B_H(ctx, aux);
           
#pragma unroll
            for ( ; ctx->c < BUF_SIZE_8 && j < NONCE_SIZE_8; ++j)
            {
                ctx->b[ctx->c++] = mes[j];
            }
        }

    //====================================================================//
    //  Finalize hash
    //====================================================================//
        DEVICE_B2B_H_LAST(ctx, aux);

#pragma unroll
        for (j = 0; j < NUM_SIZE_8; ++j)
        {
            ((uint8_t *)r)[(j & 0xFFFFFFFC) + (3 - (j & 3))]
                = (ctx->h[j >> 3] >> ((j & 7) << 3)) & 0xFF;
        }

    //===================================================================//
    //  Generate indices
    //===================================================================//
#pragma unroll
        for (int i = 1; i < INDEX_SIZE_8; ++i)
        {
            ((uint8_t *)r)[NUM_SIZE_8 + i] = ((uint8_t *)r)[i];
        }

#pragma unroll
        for (int k = 0; k < K_LEN; k += INDEX_SIZE_8) 
        { 
            ind[k] = r[k >> 2] & N_MASK; 
        
#pragma unroll 
            for (int i = 1; i < INDEX_SIZE_8; ++i) 
            { 
                ind[k + i] 
                    = (
                        (r[k >> 2] << (i << 3))
                        | (r[(k >> 2) + 1] >> (32 - (i << 3)))
                    ) & N_MASK; 
            } 
        } 

    //===================================================================//
    //  Calculate result
    //===================================================================//
        // first addition of hashes -> r
        asm volatile (
            "add.cc.u32 %0, %1, %2;":
            "=r"(r[0]): "r"(hash[ind[0] << 3]), "r"(hash[ind[1] << 3])
        );

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile (
                "addc.cc.u32 %0, %1, %2;":
                "=r"(r[i]):
                "r"(hash[(ind[0] << 3) + i]), "r"(hash[(ind[1] << 3) + i])
            );
        }

        asm volatile ("addc.u32 %0, 0, 0;": "=r"(r[8]));

     // remaining additions
#pragma unroll
        for (int k = 2; k < K_LEN; ++k)
        {
            asm volatile (
                "add.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(hash[ind[k] << 3])
            );

#pragma unroll
            for (int i = 1; i < 8; ++i)
            {
                asm volatile (
                    "addc.cc.u32 %0, %0, %1;":
                    "+r"(r[i]): "r"(hash[(ind[k] << 3) + i])
                );
            }

            asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[8]));
        }

        // subtraction of secret key
        asm volatile ("sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(sk[0]));

#pragma unroll
        for (int i = 1; i < 8; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(sk[i]));
        }

        asm volatile ("subc.u32 %0, %0, 0;": "+r"(r[8]));

    //===================================================================//
    //  Result mod Q
    //===================================================================//
        // 20 bytes
        uint32_t * med = ind;
        // 4 bytes
        uint32_t * d = ind + 5; 
        uint32_t * carry = d;

        d[0] = r[8];

    //====================================================================//
        asm volatile ("mul.lo.u32 %0, %1, 0xD0364141;": "=r"(med[0]): "r"(*d));
        asm volatile ("mul.hi.u32 %0, %1, 0xD0364141;": "=r"(med[1]): "r"(*d));
        asm volatile ("mul.lo.u32 %0, %1, 0xAF48A03B;": "=r"(med[2]): "r"(*d));
        asm volatile ("mul.hi.u32 %0, %1, 0xAF48A03B;": "=r"(med[3]): "r"(*d));

        asm volatile (
            "mad.lo.cc.u32 %0, %1, 0xBFD25E8C, %0;": "+r"(med[1]): "r"(*d)
        );

        asm volatile (
            "madc.hi.cc.u32 %0, %1, 0xBFD25E8C, %0;": "+r"(med[2]): "r"(*d)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xBAAEDCE6, %0;": "+r"(med[3]): "r"(*d)
        );

        asm volatile ("madc.hi.u32 %0, %1, 0xBAAEDCE6, 0;": "=r"(med[4]): "r"(*d));

    //====================================================================//
        asm volatile ("sub.cc.u32 %0, %0, %1;": "+r"(r[0]): "r"(med[0]));

#pragma unroll
        for (int i = 1; i < 5; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, %1;": "+r"(r[i]): "r"(med[i]));
        }

#pragma unroll
        for (int i = 5; i < 7; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, 0;": "+r"(r[i]));
        }

        asm volatile ("subc.u32 %0, %0, 0;": "+r"(r[7]));

    //====================================================================//
        d[1] = d[0] >> 31;
        d[0] <<= 1;

        asm volatile ("add.cc.u32 %0, %0, %1;": "+r"(r[4]): "r"(d[0]));
        asm volatile ("addc.cc.u32 %0, %0, %1;": "+r"(r[5]): "r"(d[1]));
        asm volatile ("addc.cc.u32 %0, %0, 0;": "+r"(r[6]));
        asm volatile ("addc.u32 %0, %0, 0;": "+r"(r[7]));

    //====================================================================//
        asm volatile ("sub.cc.u32 %0, %0, 0xD0364141;": "+r"(r[0]));
        asm volatile ("subc.cc.u32 %0, %0, 0xBFD25E8C;": "+r"(r[1]));
        asm volatile ("subc.cc.u32 %0, %0, 0xAF48A03B;": "+r"(r[2]));
        asm volatile ("subc.cc.u32 %0, %0, 0xBAAEDCE6;": "+r"(r[3]));
        asm volatile ("subc.cc.u32 %0, %0, 0xFFFFFFFE;": "+r"(r[4]));

#pragma unroll
        for (int i = 5; i < 8; ++i)
        {
            asm volatile ("subc.cc.u32 %0, %0, 0xFFFFFFFF;": "+r"(r[i]));
        }

        asm volatile ("subc.u32 %0, 0, 0;": "=r"(*carry));

        *carry = 0 - *carry;

    //====================================================================//
        asm volatile (
            "mad.lo.cc.u32 %0, %1, 0xD0364141, %0;": "+r"(r[0]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xBFD25E8C, %0;": "+r"(r[1]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xAF48A03B, %0;": "+r"(r[2]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xBAAEDCE6, %0;": "+r"(r[3]): "r"(*carry)
        );

        asm volatile (
            "madc.lo.cc.u32 %0, %1, 0xFFFFFFFE, %0;": "+r"(r[4]): "r"(*carry)
        );

#pragma unroll
        for (int i = 5; i < 7; ++i)
        {
            asm volatile (
                "madc.lo.cc.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(r[i]): "r"(*carry)
            );
        }

        asm volatile (
            "madc.lo.u32 %0, %1, 0xFFFFFFFF, %0;": "+r"(r[7]): "r"(*carry)
        );

    //===================================================================//
    //  Dump result to global memory -- LITTLE ENDIAN
    //===================================================================//
        j = ((uint64_t *)r)[3] < ((uint64_t *)bound)[3]
            || ((uint64_t *)r)[3] == ((uint64_t *)bound)[3] && (
                ((uint64_t *)r)[2] < ((uint64_t *)bound)[2]
                || ((uint64_t *)r)[2] == ((uint64_t *)bound)[2] && (
                    ((uint64_t *)r)[1] < ((uint64_t *)bound)[1]
                    || ((uint64_t *)r)[1] == ((uint64_t *)bound)[1]
                    && ((uint64_t *)r)[0] < ((uint64_t *)bound)[0]
                )
            );

        valid[tid] = (1 - !j) * (tid + 1);

#pragma unroll
        for (int i = 0; i < NUM_SIZE_32; ++i)
        {
            res[tid * NUM_SIZE_32 + i] = r[i];
        }

        __syncthreads();
    }

    return;
}

// mining.cu
